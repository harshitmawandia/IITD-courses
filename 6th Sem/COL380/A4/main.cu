#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16
#define NUM_STREAMS 2

__global__ void sparse_matrix_mult(int *d_row_ptr, int *d_col_ind, short *d_val, short *d_A, short *d_B, short *d_C, int n)
{
    __shared__ short shared_A[TILE_SIZE][TILE_SIZE];
    __shared__ short shared_B[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    short sum = 0;
    int start = d_row_ptr[row];
    int end = d_row_ptr[row+1];

    for (int k = start; k < end; k++) {
        int idx = d_col_ind[k];
        if (idx == col) {
            sum += d_val[k] * d_B[row * n + idx];
        }
        else {
            shared_A[ty][tx] = d_val[k];
            shared_B[ty][tx] = d_B[idx * n + col];
            __syncthreads();
            for (int i = 0; i < TILE_SIZE; i++) {
                sum += shared_A[ty][i] * shared_B[i][tx];
            }
            __syncthreads();
        }
    }

    d_C[row * n + col] = sum;
}

int main(int argc, char** argv)
{
    if (argc < 2) {
        printf("Usage: %s matrix.bin\n", argv[0]);
        exit(1);
    }

    char *filename = argv[1];
    FILE *fp = fopen(filename, "rb");

    if (fp == NULL) {
        printf("Error: could not open file %s\n", filename);
        exit(1);
    }

    int n;
    int num_blocks;
    int num_nonzeros;
    int block_size;
    int *row_ptr;
    int *col_ind;
    short *val;

    fread(&n, sizeof(int), 1, fp);
    fread(&num_blocks, sizeof(int), 1, fp);
    fread(&num_nonzeros, sizeof(int), 1, fp);
    fread(&block_size, sizeof(int), 1, fp);

    row_ptr = (int*)malloc((n+1) * sizeof(int));
    col_ind = (int*)malloc(num_nonzeros * sizeof(int));
    val = (short*)malloc(num_nonzeros * sizeof(short));

    fread(row_ptr, sizeof(int), n+1, fp);
    fread(col_ind, sizeof(int), num_nonzeros, fp);
    fread(val, sizeof(short), num_nonzeros, fp);

    fclose(fp);

    short *h_A = (short*)malloc(n * n * sizeof(short));
    short *h_B = (short*)malloc(n * n * sizeof(short));
    short *h_C = (short*)malloc(n * n * sizeof(short));

    for (int i = 0; i < n * n; i++) {
        h_A[i] = 0;
        h_B[i] = 0;
        h_C[i] = 0;
    }

    for (int i = 0; i < n; i += block_size) {
        for (int j = 0; j < n; j +=block_size) {
            int index = row_ptr[i / block_size];
            int next_index = row_ptr[(i / block_size) + 1];
            for (int k = index; k < next_index; k++) {
                int row = col_ind[k];
                int col = j + block_size - 1;
                if (col >= n) {
                    col = n - 1;
                }
                for (int r = i; r < i + block_size; r++) {
                    for (int c = j; c <= col; c++) {
                        h_A[r * n + c] = val[k];
                    }
                }
            }
        }
    }
    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimGrid(ceil((float)n/TILE_SIZE), ceil((float)n/TILE_SIZE), 1);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamCreate(&streams[i]);
    }

    int chunk_size = n / NUM_STREAMS;
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * chunk_size * n;
        int size = chunk_size * n;
        hipMemcpyAsync(&d_B[offset], &h_B[offset], size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(&d_C[offset], &h_C[offset], size * sizeof(float), hipMemcpyHostToDevice, streams[i]);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * chunk_size * n;
        sparse_matrix_mult<<<dimGrid, dimBlock, 0, streams[i]>>>(d_row_ptr, d_col_ind, d_val, d_A, &d_B[offset], &d_C[offset], n);
    }

    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = i * chunk_size * n;
        int size = chunk_size * n;
        hipMemcpyAsync(&h_C[offset], &d_C[offset], size * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
    }

    hipDeviceSynchronize();

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamDestroy(streams[i]);
    }

    free(h_A);
    free(h_B);
    free(h_C);
    free(row_ptr);
    free(col_ind);
    free(val);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_row_ptr);
    hipFree(d_col_ind);
    hipFree(d_val);

    return 0;

}
